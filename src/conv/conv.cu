#include "hip/hip_runtime.h"
//
// conv.cu
//
// Part of ENGI-9869 Assignment #5
//
// Author: Zhen Guan
// Email: zguan@mun.ca
// Student Number: 202191382
//

#include <stdexcept>
#include <chrono>

#include "conv/conv.hpp"
#include "util/array.hpp"

// Convolution on a single pixel.
// For a NxN picture, there bill be NxN cells running simultaneously.
__global__ void device_convolution_cell(
    const int *kernel_2d,
    const int kernel_size,
    const unsigned char *input_image_2d,
    unsigned char *output_image_2d,
    const int image_width,
    const int image_height,
    const int num_workers_sqrt,
    const int greyscale_max
) {
    const int my_rank = (int) threadIdx.x;
    const int block_size = image_width / num_workers_sqrt;
    const int from_row_index = (my_rank / num_workers_sqrt) * block_size;
    const int to_row_index = from_row_index + block_size;
    const int from_col_index = (my_rank % num_workers_sqrt) * block_size;
    const int to_col_index = from_col_index + block_size;

    for (int i = from_row_index; i < to_row_index; ++i) {
        for (int j = from_col_index; j < to_col_index; ++j) {
            // Convolution at input_image_2d[i][j];
            int sum = 0;
            for (int x = 0; x < kernel_size; ++x) {
                for (int y = 0; y < kernel_size; ++y) {
                    int row_index = i + x - kernel_size / 2;
                    int col_index = j + y - kernel_size / 2;
                    if (row_index < 0 || row_index >= image_height
                        || col_index < 0 || col_index >= image_width) {
                        // greyscale at outside the image (e.g. minus index) is zero.
                        sum += 0;
                    }
                    else {
                        sum += ARRAY2D_AT(kernel_2d, kernel_size, x, y)
                            * (int)ARRAY2D_AT(input_image_2d, image_width, row_index, col_index);
                    }
                }
            }
            if (sum < 0) {
                sum = 0;
            }
            else if (sum > greyscale_max) {
                sum = greyscale_max;
            }
            ARRAY2D_AT(output_image_2d, image_width, i, j) = (unsigned char)sum;
        }
    }

}

static void assertCudaError(hipError_t error) {
    if (error != hipSuccess) {
        throw std::runtime_error("CUDA error: " + std::string(hipGetErrorString(error)));
    }
}

// Check if an integer has an integer square root or not.
// If it has, return the integer square root;
// Otherwise, return 0.
static int sqrt_integer_only(int n) {
    if (n == 1) {
        return 1;
    }

    int n_half = n / 2;
    for (int i = 1; i <= n_half; ++i) {
        if (i * i == n) {
            return i;
        }
    }
    return 0;
}

// Convolute a square block of the image with the given kernel.
// Points outside the block but still inside the image, will be set correctly;
// Points outside the image will be assumed to 0.
void conv::convolution(
    int num_workers,
    int kernel_2d[conv::CONV_KERNEL_SIZE * conv::CONV_KERNEL_SIZE],
    int kernel_size,
    unsigned char *input_image_2d,
    unsigned char *output_image_2d,
    int image_width,
    int image_height,
    int greyscale_max
) {
    // Check if the image is square.
    if (image_width != image_height) {
        throw std::runtime_error("Image is not square.");
    }

    // Check if the num_workers is a square number.
    const int num_workers_sqrt = sqrt_integer_only(num_workers);
    if (num_workers_sqrt == 0) {
        throw std::runtime_error("num_workers is not a square number.");
    }

    // const int num_block_cells = image_width * image_height / num_workers;

    // Prepare device memory.
    int *kernel_2d_device;
    unsigned char *input_image_2d_device;
    unsigned char *output_image_2d_device;

    assertCudaError(hipMalloc(&kernel_2d_device, sizeof(int) * kernel_size * kernel_size));
    assertCudaError(hipMalloc(&input_image_2d_device, sizeof(unsigned char) * image_width * image_height));
    assertCudaError(hipMalloc(&output_image_2d_device, sizeof(unsigned char) * image_width * image_height));

    // Prepare inputs.
    assertCudaError(hipMemcpy(kernel_2d_device, kernel_2d, sizeof(int) * kernel_size * kernel_size, hipMemcpyHostToDevice));
    assertCudaError(hipMemcpy(input_image_2d_device, input_image_2d, sizeof(unsigned char) * image_width * image_height, hipMemcpyHostToDevice));

    // Launch the kernel.

    float time;
    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    device_convolution_cell<<<1, num_workers>>>(
        kernel_2d_device,
        kernel_size,
        input_image_2d_device,
        output_image_2d_device,
        image_width,
        image_height,
        num_workers_sqrt,
        greyscale_max
    );

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("Time elapsed: %.0fms\n", time);

    // Copy output.
    assertCudaError(hipMemcpy(output_image_2d, output_image_2d_device, sizeof(unsigned char) * image_width * image_height, hipMemcpyDeviceToHost));
    assertCudaError(hipFree(kernel_2d_device));
    assertCudaError(hipFree(input_image_2d_device));
    assertCudaError(hipFree(output_image_2d_device));
}
